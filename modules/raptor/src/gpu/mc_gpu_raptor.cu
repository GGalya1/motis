#include "hip/hip_runtime.h"
#include "motis/raptor/gpu/mc_gpu_raptor.cuh"

#include <iostream>

#include "motis/raptor/gpu/cuda_util.h"
#include "motis/raptor/gpu/gpu_mark_store.cuh"
#include "motis/raptor/gpu/raptor_utils.cuh"
#include "motis/raptor/gpu/update_arrivals.cuh"

#include "motis/raptor/criteria/configs.h"

#include "hip/hip_cooperative_groups.h"

namespace motis::raptor {

using namespace cooperative_groups;

// leader type must be unsigned 32bit
// no leader is a zero ballot vote (all 0) minus 1 => with underflow all 1's
constexpr unsigned int FULL_MASK = 0xFFFFffff;
constexpr unsigned int NO_LEADER = FULL_MASK;

template <typename CriteriaConfig>
__device__ dimension_id
get_moc(typename CriteriaConfig::CriteriaData const& d) {
  return 3;
}

template <>
__device__ dimension_id
get_moc<MaxOccupancy>(MaxOccupancy ::CriteriaData const& d) {
  return d.max_occupancy_;
}

template <>
__device__ dimension_id
get_moc<TimeSlottedOccupancy>(TimeSlottedOccupancy::CriteriaData const& d) {
  return d.initial_soc_idx_ + d.occ_time_slot_;
}

template <typename CriteriaConfig>
__device__ occ_t
get_initial_moc(typename CriteriaConfig::CriteriaData const& d) {
  return 255;
}

template <>
__device__ occ_t
get_initial_moc<MaxOccupancy>(MaxOccupancy ::CriteriaData const& d) {
  return d.initial_moc_idx_;
}

template <>
__device__ occ_t get_initial_moc<TimeSlottedOccupancy>(
    TimeSlottedOccupancy::CriteriaData const& d) {
  return d.summed_occ_time_;
}

template <typename CriteriaConfig>
__device__ void mc_copy_marked_arrivals(time* const to, time const* const from,
                                        unsigned int* station_marks,
                                        device_gpu_timetable const& tt) {
  auto const global_stride = get_global_stride();

  auto arr_idx = get_global_thread_id();
  auto trait_size = CriteriaConfig::trait_size();
  auto max_arrival_idx = tt.stop_count_ * trait_size;
  for (; arr_idx < max_arrival_idx; arr_idx += global_stride) {
    // auto const stop_id = arr_idx / trait_size;

    // only copy the values for station + trait offset which are valid
    if (marked(station_marks, arr_idx) && valid(from[arr_idx])) {
      to[arr_idx] = from[arr_idx];
    } else {
      to[arr_idx] = invalid<time>;
    }
  }
}

template <typename CriteriaConfig>
__device__ void mc_copy_and_min_arrivals(time* const to, time* const from,
                                         device_gpu_timetable const& tt) {
  auto const global_stride = get_global_stride();

  auto arr_idx = get_global_thread_id();
  auto const max_arr_idx = tt.stop_count_ * CriteriaConfig::trait_size();
  for (; arr_idx < max_arr_idx; arr_idx += global_stride) {
    to[arr_idx] = min(from[arr_idx], to[arr_idx]);
  }
}

__device__ __forceinline__ unsigned get_criteria_propagation_mask(
    unsigned const leader, unsigned const stop_count) {
  auto const stops_to_update = stop_count - leader - 1;
  auto const mask = (1 << stops_to_update) - 1;
  return (mask << (leader + 1));
}

__device__ __forceinline__ unsigned get_last_departure_stop(
    unsigned criteria_mask) {
  unsigned const rev = __brev(criteria_mask);
  unsigned const last = 32 - __ffs(rev);
  return last;
}

template <typename CriteriaConfig>
__device__ __forceinline__ time get_earliest_arrival(
    const time* const earliest_arrivals, stop_id const target_stop_id,
    stop_id const current_stop_id, trait_id const write_to_offset) {

  arrival_id const current_idx =
      CriteriaConfig::get_arrival_idx(current_stop_id, write_to_offset);
  time const stop_ea = earliest_arrivals[current_idx];
  auto const target_arr_idx =
      CriteriaConfig::get_arrival_idx(target_stop_id, write_to_offset);
  time const target_ea = earliest_arrivals[target_arr_idx];
  return umin(stop_ea, target_ea);
}

template <typename CriteriaConfig>
__device__ void mc_update_route_larger32(
    route_id const r_id, gpu_route const route, trait_id const t_offset,
    time const* const prev_arrivals, time* const arrivals,
    time* const earliest_arrivals, stop_id const target_stop_id,
    uint32_t* station_marks, device_gpu_timetable const& tt) {

  auto const t_id = threadIdx.x;

  stop_id stop_id_t = invalid<stop_id>;
  time prev_arrival = invalid<time>;
  time stop_arrival = invalid<time>;
  time stop_departure = invalid<time>;

  typename CriteriaConfig::CriteriaData aggregate{};
  unsigned last_known_dep_stop = invalid<unsigned>;

  int active_stop_count = route.stop_count_;

  // this is ceil(stop_count / 32)
  int const stage_count = (route.stop_count_ + (32 - 1)) >> 5;
  int active_stage_count = stage_count;

  unsigned int leader = NO_LEADER;
  unsigned int any_arrival = 0;
  unsigned int criteria_mask = 0;

  for (int trip_offset = 0; trip_offset < route.trip_count_; ++trip_offset) {
    CriteriaConfig::reset_traits_aggregate(aggregate, r_id, trip_offset,
                                           t_offset);

    //    if (t_id == 0 && r_id == 31368 && t_offset == 0) {
    //      printf(
    //          "r_id %i;\tscanning trip_offset %i;\tactive stage count:
    //          %i;\tactive " "stop count %i\n", r_id, trip_offset,
    //          active_stage_count, active_stop_count);
    //    }

    for (uint32_t current_stage = 0; current_stage < active_stage_count;
         ++current_stage) {

      uint32_t stage_id = (current_stage << 5) + t_id;  // stage_id ^= stop_id

      // load the prev arrivals for the current stage
      if (stage_id < active_stop_count) {
        stop_id_t = tt.route_stops_[route.index_to_route_stops_ + stage_id];
        auto const stop_arr_idx =
            CriteriaConfig::get_arrival_idx(stop_id_t, t_offset);
        prev_arrival = prev_arrivals[stop_arr_idx];
      }

      any_arrival |= __any_sync(FULL_MASK, valid(prev_arrival));
      //      if (t_id == 0 && r_id == 31368 && t_offset == 0 && current_stage
      //      == 0) {
      //        printf(
      //            "r_id %i;\tscanning trip_offset %i;\tprev arr: %i;\tany arr
      //            %i\n", r_id, trip_offset, prev_arrival, any_arrival);
      //      }

      if (current_stage == active_stage_count - 1 && !any_arrival) {
        return;
      }

      if (!any_arrival) {
        continue;
      }

      // load the stop times for the current stage
      if (stage_id < active_stop_count) {
        auto const st_idx = route.index_to_stop_times_ +
                            (trip_offset * route.stop_count_) + stage_id;
        stop_departure = tt.stop_departures_[st_idx];
      }

      // get the current stage leader
      unsigned int ballot = __ballot_sync(
          FULL_MASK, (stage_id < active_stop_count) && valid(prev_arrival) &&
                         valid(stop_departure) &&
                         (prev_arrival <= stop_departure));

      // index of first possible departure station on this stage
      leader = __ffs(ballot) - 1;

      auto const stop_count_on_stage =
          active_stop_count < ((current_stage + 1) << 5)
              ? (active_stop_count - (current_stage << 5))
              : 32;

      //      if ((r_id == 31368) && (trip_offset == 17 || trip_offset == 15) &&
      //          t_offset == 0 && t_id < stop_count_on_stage)
      //        printf(
      //            "Ballot Mask (%i) for r_id: %i, t_offset: %i, trip_offset: "
      //            "%i;\tstage: %i;\tleader: %i;\t%x\n",
      //            t_id, r_id, t_offset, trip_offset, current_stage, leader,
      //            ballot);

      if (valid(last_known_dep_stop)) {
        // no leader in current stags; though maybe there is a leader in the
        //   previous stage
        //        printf("Active Stop count %i;\ton stage: %i\n",
        //        stop_count_on_stage, current_stage);
        criteria_mask = (1 << stop_count_on_stage) - 1;
      } else if (leader != NO_LEADER) {
        criteria_mask =
            get_criteria_propagation_mask(leader, stop_count_on_stage);
      } else {
        // no departure stop up to now
        continue;
      }

      //      if (t_id == 0 && (r_id == 118) && trip_offset == 1) {
      //        printf(
      //            "Criteria Mask for r_id: %i, t_offset: %i, trip_offset: "
      //            "%i;\tstage: %i\t%x\n",
      //            r_id, t_offset, trip_offset, current_stage,
      //            criteria_mask);
      //      }

      if (leader > 0 && valid(last_known_dep_stop)) {
        // in a case where there is no departure location in this stage
        // or the departure location (leader) is after this stop
        // and there is a valid known departure stop from teh previous stage
        // carry over the aggregate from the last stop in the previous stage
        // to the first stop in this stage
        unsigned carry_mask = (1 << 31) | 1;
        CriteriaConfig::carry_to_next_stage(carry_mask, aggregate);

        //        if (t_id == 0 && (r_id == 29933 && trip_offset == 0))
        //          printf(
        //              "t_id: %i\tr_id: %i\tt_offset: %i\ttrip_id: %i;\tstage:
        //              "
        //              "%i\tfound "
        //              "carried moc for s_id: %i\tmoc: %i;\tinitial moc: %i\n",
        //              t_id, r_id, t_offset, trip_offset, current_stage,
        //              stop_id_t, get_moc<CriteriaConfig>(aggregate),
        //              get_initial_moc<CriteriaConfig>(aggregate));
      }

      auto const has_carry_value = t_id == 0 && valid(last_known_dep_stop);
      if (!has_carry_value) {
        //        if (r_id == 29933 && current_stage == 1 && trip_offset == 0 &&
        //        t_id == 18)
        //          printf("Resetting Aggregate for t_id: %i\tstage:%i\n ",
        //          t_id,
        //                 current_stage);

        // don't reset if this has a carry value even if it is a departure
        // station
        //  as arrival time might be improved through the carry value
        CriteriaConfig::reset_traits_aggregate(aggregate, r_id, trip_offset,
                                               t_offset);
      }

      if (leader != NO_LEADER) {
        // adjust the determined departure location to the current stage
        leader += current_stage << 5;
      }

      // update this stage if there is a leader or a known dep stop from
      // one of the previous stages
      bool wrote_satis_time = false;
      if ((leader != NO_LEADER || valid(last_known_dep_stop)) &&
          stage_id < active_stop_count) {

        if ((leader != NO_LEADER && stage_id > leader) ||
            (valid(last_known_dep_stop) && stage_id > last_known_dep_stop)) {
          auto const st_idx = route.index_to_stop_times_ +
                              (trip_offset * route.stop_count_) + stage_id;
          stop_arrival = tt.stop_arrivals_[st_idx];

          // is_departure_stop is local to the current stage
          auto const is_departure_stop = (((1 << t_id) & ballot) >> t_id);

          //          if (is_departure_stop && (r_id == 31368) && t_offset == 0
          //          &&
          //              (current_stage == 0) &&
          //              (trip_offset == 17 || trip_offset == 15)) {
          //            printf(
          //                "Is Departure Stop: r_id: %i\tt_offset:
          //                %i;\ttrip_id: % i "
          //                "\tt_id %i\ts_id: %i;\tstage: %i\n",
          //                r_id, t_offset, trip_offset, t_id, stop_id_t,
          //                current_stage);
          //          }

          CriteriaConfig::update_traits_aggregate(aggregate, tt, prev_arrivals,
                                                  t_id, st_idx);

          //            if (r_id == 29933 && current_stage == 1 && trip_offset
          //            == 0) {
          //              printf(
          //                  "Updated aggregate for t_id: %i\tstage: %i;\tnew
          //                  moc: "
          //                  "%i;\tinit: %i\n",
          //                  t_id, current_stage,
          //                  get_moc<CriteriaConfig>(aggregate),
          //                  get_initial_moc<CriteriaConfig>(aggregate));
          //            }

          // propagate the additional criteria attributes
          for (uint32_t idx = __ffs(criteria_mask); idx < stop_count_on_stage;
               ++idx) {
            // internally uses __shfl_up_sync to propagate the criteria values
            //  along the traits while allowing for max/min/sum operations
            CriteriaConfig::propagate_and_merge_if_needed(
                criteria_mask, aggregate, is_departure_stop,
                idx <= t_id
                    // prevent write update if this has carry value
                    && !has_carry_value);

            //            if (r_id == 29933 && current_stage == 1 && trip_offset
            //            == 0)
            //              printf(
            //                  "Updated aggregate in Loop for t_id: %i\tstage:%
            //                  i;\tnew " "moc: %i;\tnew init %i;\tmask: %x\n",
            //                  t_id, current_stage,
            //                  get_moc<CriteriaConfig>(aggregate),
            //                  get_initial_moc<CriteriaConfig>(aggregate),
            //                  criteria_mask);
          }

          auto const write_to_offset =
              CriteriaConfig::get_write_to_trait_id(aggregate);
          //          if (r_id == 31368 && (trip_offset == 17 || trip_offset ==
          //          15) &&
          //              t_id < 4 && current_stage == 0) {
          //            printf(
          //                "\nt_id: %i\tr_id: %i\tt_offset: %i\ttrip_id:
          //                %i\tfound moc" " for s_id: %i\tmoc: %i;\twrite idx:
          //                %i;\tinitial moc: "
          //                "%i;\tarrival: %i\n",
          //                t_id, r_id, t_offset, trip_offset, stop_id_t,
          //                get_moc<CriteriaConfig>(aggregate), write_to_offset,
          //                get_initial_moc<CriteriaConfig>(aggregate),
          //                stop_arrival);
          //          }

          if (valid(write_to_offset)) {
            auto const earliest_arrival = get_earliest_arrival<CriteriaConfig>(
                earliest_arrivals, target_stop_id, stop_id_t, write_to_offset);

            if (stop_arrival < earliest_arrival) {
              auto const write_to_arr_idx =
                  CriteriaConfig::get_arrival_idx(stop_id_t, write_to_offset);
              bool updated =
                  update_arrival(arrivals, write_to_arr_idx, stop_arrival);
              if (updated) {
                wrote_satis_time =
                    CriteriaConfig::is_trait_satisfied(aggregate, t_offset);
                //                if (stop_id_t == 19386 && r_id == 29933 &&
                //                trip_offset == 0) {
                //                  printf(
                //                      "Wrote arrival to stop %i from r_id >
                //                      32: "
                //                      "%i;\ttrip_offset: "
                //                      "%i;\twrite_offset: %i;\tt_offset:
                //                      %i;\tarrival: "
                //                      "%i;\tballot: %x;\tadd: %i;\tstage:
                //                      %i;\tinit add: %i\n", stop_id_t, r_id,
                //                      trip_offset, write_to_offset, t_offset,
                //                      stop_arrival, ballot,
                //                      get_moc<CriteriaConfig>(aggregate),
                //                      current_stage,
                //                      get_initial_moc<CriteriaConfig>(aggregate));
                //                }

                update_arrival(earliest_arrivals, write_to_arr_idx,
                               stop_arrival);
                mark(station_marks, write_to_arr_idx);
              }
            }
          }
        }
      }

      if (leader != NO_LEADER) {
        if (current_stage == active_stage_count - 1) {
          // at the last stage check the stop satisfaction and reduce asc if
          // possible
          time satisfied_ea = invalid<time>;
          if ((1 << t_id) & criteria_mask) {
            auto satisfied_arr_idx =
                CriteriaConfig::get_arrival_idx(stop_id_t, t_offset);
            satisfied_ea = get_earliest_arrival<CriteriaConfig>(
                earliest_arrivals, stop_id_t, target_stop_id, t_offset);
          }

          auto satisfied_ballot = __ballot_sync(
              FULL_MASK,
              stage_id < active_stop_count &&
                  (wrote_satis_time ||
                   (valid(stop_arrival) && satisfied_ea <= stop_arrival)));

          auto stage_asc = active_stop_count - (current_stage << 5);
          auto const helper_mask =
              stage_asc < 32 ? __brev((1 << (32 - stage_asc)) - 1) : 0;
          auto const inverted_ballot = ~(satisfied_ballot | helper_mask);
          auto leading_zero_count = __clz(inverted_ballot);
          auto const init_lzc = leading_zero_count;
          leading_zero_count -= (32 - stage_asc);
          active_stop_count -= leading_zero_count;

          auto const init_stage_asc = stage_asc;
          stage_asc -= leading_zero_count;

          if (stage_asc == 0) {
            active_stage_count -= 1;
          }

          //          if (r_id == 31368 && t_offset == 0 && trip_offset == 15 &&
          //              current_stage == 0) {
          //            printf(
          //                "Recalc ASC (%i): sat ea: %i;\tsat ball %x;\tstage
          //                asc "
          //                "%i;\thelper %x;\tinv_ball %x;\tlzc %i;\tinit lzc
          //                %i;\tasc "
          //                "%i;\tnew stage asc %i\n",
          //                t_id, satisfied_ea, satisfied_ballot,
          //                init_stage_asc, helper_mask, inverted_ballot,
          //                leading_zero_count, init_lzc, active_stop_count,
          //                stage_asc);
          //          }

        } else {
          // there is a leader in the current stage; therefore safe the last
          // possible departure stop for updates to the next stage
          last_known_dep_stop = get_last_departure_stop(criteria_mask);
          last_known_dep_stop += current_stage << 5;
        }
      }
    }
  }
}

template <typename CriteriaConfig>
__device__ void mc_update_route_smaller32(
    route_id const r_id, gpu_route const route, trait_id const t_offset,
    time const* const prev_arrivals, time* const arrivals,
    time* const earliest_arrivals, stop_id const target_stop_id,
    uint32_t* station_marks, device_gpu_timetable const& tt) {

  auto const t_id = threadIdx.x;

  stop_id s_id = invalid<stop_id>;
  time prev_arrival = invalid<time>;
  time stop_arrival = invalid<time>;
  time stop_departure = invalid<time>;

  typename CriteriaConfig::CriteriaData aggregate{};

  unsigned leader = route.stop_count_;
  unsigned int active_stop_count = route.stop_count_;

  if (t_id < active_stop_count) {
    s_id = tt.route_stops_[route.index_to_route_stops_ + t_id];
    auto const stop_arr_idx = CriteriaConfig::get_arrival_idx(s_id, t_offset);
    prev_arrival = prev_arrivals[stop_arr_idx];
  }

  // we skip updates if there is no feasible departure station
  //  on this route with the given trait offset
  if (!__any_sync(FULL_MASK, valid(prev_arrivals))) {
    return;
  }

  for (trip_id trip_offset = 0; trip_offset < route.trip_count_;
       ++trip_offset) {
    CriteriaConfig::reset_traits_aggregate(aggregate, r_id, trip_offset,
                                           t_offset);

    if (t_id < active_stop_count) {
      auto const st_index =
          route.index_to_stop_times_ + (trip_offset * route.stop_count_) + t_id;
      stop_departure = tt.stop_departures_[st_index];
    }

    unsigned ballot = __ballot_sync(
        FULL_MASK, (t_id < active_stop_count) && valid(prev_arrival) &&
                       valid(stop_departure) &&
                       (prev_arrival <= stop_departure));

    // index of the first departure location on route
    leader = __ffs(ballot) - 1;

    //    if ((r_id == 14994) && t_offset == 0 && trip_offset == 1 && t_id <
    //    route.stop_count_)
    //      printf(
    //          "Ballot Mask (%i) for r_id: %i, t_offset: %i, trip_offset: "
    //          "%i;\t%x;\tasc: %i;\tprev_arr: %i;\tdeparture: %i;\tsmaller asc:
    //          %i;\tvalid pa %i;\tvalid dep: %i;\tpa <= dep: %i;\tleader %i\n",
    //          t_id, r_id, t_offset, trip_offset, ballot, active_stop_count,
    //          prev_arrival, stop_departure, t_id < active_stop_count,
    //          valid(prev_arrival), valid(stop_departure), prev_arrival <=
    //          stop_departure, leader);

    if (leader == NO_LEADER) continue;  // No feasible departure on this trip

    unsigned criteria_mask =
        get_criteria_propagation_mask(leader, active_stop_count);

    //    if (t_id == 0 && (r_id == 20530) && t_offset == 0 && trip_offset == 1)
    //    {
    //      printf(
    //          "Criteria Mask for r_id: %i, t_offset: %i, trip_offset: "
    //          "%i;\t%x\n",
    //          r_id, t_offset, trip_offset, criteria_mask);
    //    }

    bool wrote_satis_time = false;
    if (t_id > leader && t_id < active_stop_count) {
      auto const st_index =
          route.index_to_stop_times_ + (trip_offset * route.stop_count_) + t_id;

      stop_arrival = tt.stop_arrivals_[st_index];
      auto const is_departure_stop = (((1 << t_id) & ballot) >> t_id);

      //      if (is_departure_stop && (r_id == 14994) && t_offset == 0 &&
      //      trip_offset == 1)
      //        printf(
      //            "Is Departure Stop: r_id: %i\tt_offset: %i;\t trip_id:
      //            %i\tt_id: "
      //            "%i\ts_id: %i\n",
      //            r_id, t_offset, trip_offset, t_id, s_id);

      CriteriaConfig::update_traits_aggregate(
          aggregate, tt, prev_arrivals, t_id /* == stop offset */, st_index);

      // propagate the additional criteria attributes
      for (uint32_t idx = leader + 1; idx < active_stop_count; ++idx) {
        // internally uses __shfl_up_sync to propagate the criteria values
        //  along the traits while allowing for max/min/sum operations
        CriteriaConfig::propagate_and_merge_if_needed(
            criteria_mask, aggregate, is_departure_stop, idx <= t_id);
      }

      auto const write_to_offset =
          CriteriaConfig::get_write_to_trait_id(aggregate);

      //      if(r_id == 14994 && t_offset == 0 && trip_offset == 1)
      //        printf("Received (%i) write offset: %i\n", t_id,
      //        write_to_offset);

      if (valid(write_to_offset) && t_id > leader) {

        // Note: Earliest Arrival may, when reaching this point not be the
        //       'earliest arrival' at this stop, but it gives a sufficient
        //       upper bound and allows preventing arrival time which are the
        //       same as for one round earlier
        auto const earliest_arrival = get_earliest_arrival<CriteriaConfig>(
            earliest_arrivals, target_stop_id, s_id, write_to_offset);

        //          if (r_id == 14994 && t_offset == 0 && trip_offset == 1)
        //            printf(
        //                "\nt_id: %i\tr_id: %i\tt_offset: %i\ttrip_id:
        //                %i\tfound moc" " for s_id: %i\tmoc: %i;\twrite idx:
        //                %i;\tinitial moc: "
        //                "%i;\tarrival: %i;\tballot: %x;\tasc: %i;\n",
        //                t_id, r_id, t_offset, trip_offset, s_id,
        //                get_moc<CriteriaConfig>(aggregate), write_to_offset,
        //                get_initial_moc<CriteriaConfig>(aggregate),
        //                stop_arrival, ballot, active_stop_count);

        if (stop_arrival < earliest_arrival) {
          auto const write_to_idx =
              CriteriaConfig::get_arrival_idx(s_id, write_to_offset);
          bool updated = update_arrival(arrivals, write_to_idx, stop_arrival);
          if (updated) {
            wrote_satis_time =
                CriteriaConfig::is_trait_satisfied(aggregate, t_offset);
            //            if (s_id == 19386 && (write_to_offset == 0)) {
            //              printf(
            //                  "Wrote arrival to Stop %i from r_id:
            //                  %i;\ttrip_offset:"
            //                  "%i;\twrite_offset: %i;\tt_offset %i;\tballot
            //                  mask:"
            //                  "%x\tarrival: "
            //                  "%i;\tearliest_arrivals: %i;\tadd: %i\n",
            //                  s_id, r_id, trip_offset, write_to_offset,
            //                  t_offset, ballot, stop_arrival,
            //                  earliest_arrival,
            //                  get_moc<CriteriaConfig>(aggregate));
            //            }

            update_arrival(earliest_arrivals, write_to_idx, stop_arrival);
            //          if ((r_id == 62 || r_id == 69))
            //            printf(
            //                "\nt_id: %i\tr_id: %i\tt_offset: %i\ttrip_id:
            //                %i\twrite update" "for " "s_id: %i\tto arr idx:
            //                %i\tarr_time: %i\n", t_id, r_id, t_offset,
            //                trip_offset, s_id, stop_arr_idx, stop_arrival);
            mark(station_marks, write_to_idx);
          }
        }
      }
    }

    // check if stops on route are satisfied
    if (leader != NO_LEADER) {
      time satisfied_ea = invalid<time>;
      if ((1 << t_id) & criteria_mask) {
        auto satisfied_arr_idx =
            CriteriaConfig::get_arrival_idx(s_id, t_offset);
        satisfied_ea = get_earliest_arrival<CriteriaConfig>(
            earliest_arrivals, s_id, target_stop_id, t_offset);
      }

      auto const satisfied_ballot = __ballot_sync(
          FULL_MASK, t_id < active_stop_count &&
                         (wrote_satis_time || (valid(stop_arrival) &&
                                               satisfied_ea <= stop_arrival)));

      //      if (r_id == 17290 && t_offset == 0) {
      //        printf(
      //            "t_id (%i) < asc (%i): %s;\tis_sat &&
      //            valid(arr): %s;\tsat_ea <= arr: "
      //            "%s;\tsat_ea: %i;\tarr: %i\n",
      //            t_id, active_stop_count,
      //            (t_id < active_stop_count) ? "true" :
      //            "false", (valid(satisfied_ea) &&
      //             CriteriaConfig::is_trait_satisfied(aggregate,
      //             t_offset))
      //                ? "true"
      //                : "false",
      //            (satisfied_ea <= stop_arrival &&
      //            valid(stop_arrival)) ? "true"
      //                                                                  :
      //                                                                  "false",
      //            satisfied_ea, stop_arrival);
      //      }

      // not satisfied yet but there's a chance we can reduce
      // the number of stops to be scanned on the next trip
      auto const helper_mask = __brev((1 << (32 - active_stop_count)) - 1);
      auto inverted_ballot = ~(satisfied_ballot | helper_mask);
      auto leading_zero_count = __clz(inverted_ballot);
      //      auto const initial_clz = leading_zero_count;
      leading_zero_count -= (32 - active_stop_count);

      //      auto const initial_acs = active_stop_count;
      active_stop_count -= leading_zero_count;

      //      if (r_id == 17290 && t_offset == 0 && t_id == 17)
      //      {
      //        printf(
      //            "ACS Update; sat ballot: %x;\tsat ea: "
      //            "%i;\tHM: %x;\tinv ballot: %x;\tinit clz:
      //            %i;\tclz: %i;\tinit acs: "
      //            "%i;\tnew acs: %i\n",
      //            satisfied_ballot, satisfied_ea,
      //            helper_mask, inverted_ballot, initial_clz,
      //            leading_zero_count, initial_acs,
      //            active_stop_count);
      //      }

      // if every stop is satisfied we can skip further updates
      if ((1 << route.stop_count_) - 1 == satisfied_ballot) {
        //        if (r_id == 17290 && t_offset == 0 && t_id ==
        //        17)
        //          printf("broke on trip_offset: %i",
        //          trip_offset);
        break;
      }
    }
    leader = NO_LEADER;
  }
}

template <typename CriteriaConfig>
__device__ void mc_update_footpaths_dev_scratch(
    time const* const read_arrivals, time* const write_arrivals,
    time* const earliest_arrivals, stop_id const target_stop_id,
    uint32_t* station_marks, device_gpu_timetable const& tt) {

  auto const global_stride = get_global_stride();

  auto arrival_idx = get_global_thread_id();
  auto const trait_size = CriteriaConfig::trait_size();
  auto const max_arr_idx = tt.footpath_count_ * trait_size;
  auto const target_arr_idx = CriteriaConfig::get_arrival_idx(target_stop_id);

  for (; arrival_idx < max_arr_idx; arrival_idx += global_stride) {
    auto const foot_idx = arrival_idx / trait_size;
    auto const t_offset = arrival_idx % trait_size;

    auto const footpath = tt.footpaths_[foot_idx];

    auto const from_arrival_idx =
        CriteriaConfig::get_arrival_idx(footpath.from_, t_offset);
    auto const to_arrival_idx =
        CriteriaConfig::get_arrival_idx(footpath.to_, t_offset);

    time const from_arrival = read_arrivals[from_arrival_idx];
    time const new_arrival = from_arrival + footpath.duration_;

    // this give potentially just an upper bound and not the real
    //  earliest arrival value at the time the update is written
    time const to_stop_ea = earliest_arrivals[to_arrival_idx];
    time const target_ea = earliest_arrivals[target_arr_idx];
    time const earliest_arrival = umin(to_stop_ea, target_ea);

    if (valid(from_arrival) && marked(station_marks, from_arrival_idx) &&
        new_arrival < earliest_arrival) {
      bool updated =
          update_arrival(write_arrivals, to_arrival_idx, new_arrival);
      if (updated) {
        //        if (footpath.to_ == 19386 && (t_offset == 0)) {
        //          printf(
        //              "Wrote arrival to Stop %i from footpath: from
        //              %i;\tt_offset: "
        //              "%i;\tarrival at src: %i;\tFP duration: %i\n",
        //              footpath.to_, footpath.from_, t_offset, from_arrival,
        //              footpath.duration_);
        //        }
        update_arrival(earliest_arrivals, to_arrival_idx, new_arrival);
        mark(station_marks, to_arrival_idx);
      }
    }
  }
}

template <typename CriteriaConfig>
__device__ void mc_clear_dominated_arrivals(stop_id const stop_count,
                                            time* const arrivals,
                                            time* const ea,
                                            uint32_t* station_marks) {
  auto const global_stride = get_global_stride();

  auto s_id = get_global_thread_id();
  auto trait_size = CriteriaConfig::trait_size();
  for (; s_id < stop_count; s_id += global_stride) {
    time min_arrival_at_stop = arrivals[trait_size * s_id];
//    auto unmarked = false;
    for (trait_id t_offset = 1; t_offset < trait_size; ++t_offset) {
      // if the value is larger or equal than the minimum we can prune it
      //   because it is dominated by the minimum on the earliest trait offset
      auto const arr_time = arrivals[trait_size * s_id + t_offset];
      if (valid(arr_time) && min_arrival_at_stop <= arr_time) {
//        if(!unmarked) {
//          unmarked = true;
//          printf("unmarking s_id %i;\t", s_id);
//        }
//        printf("%i;\t", t_offset);

        arrivals[trait_size * s_id + t_offset] = invalid<time>;
        unmark(station_marks, trait_size * s_id + t_offset);
        if (min_arrival_at_stop < ea[trait_size * s_id + t_offset])
          ea[trait_size * s_id + t_offset] = min_arrival_at_stop;
      } else if(arr_time < min_arrival_at_stop) {
        // a higher t_offset has a better value; remember the larger value
        //  to again check higher t_offsets against it
        min_arrival_at_stop = arrivals[trait_size * s_id + t_offset];
      }
    }

//    if (unmarked) printf("\n");
  }
}

template <typename CriteriaConfig>
__device__ void mc_update_routes_dev(time const* const prev_arrivals,
                                     time* const arrivals,
                                     time* const earliest_arrivals,
                                     uint32_t* station_marks,
                                     uint32_t* route_marks,
                                     stop_id const target_stop_id,
                                     device_gpu_timetable const& tt) {

  // blockDim.x = 32; blockDim.y = 32; gridDim.x =
  // 6; => Stride = 32*6 => 192
  auto const stride = blockDim.y * gridDim.x;
  // threadIdx.y = 1..32 + (blockDim.y = 32 * blockIdx.x = 1..6)
  auto const start_idx = threadIdx.y + (blockDim.y * blockIdx.x);

  auto const trait_size = CriteriaConfig::trait_size();
  auto const max_idx = tt.route_count_ * trait_size;
  for (auto idx = start_idx; idx < max_idx; idx += stride) {
    if (!marked(route_marks, idx)) {
      continue;
    }

    auto const r_id = idx / trait_size;
    auto const route = tt.routes_[r_id];
    auto const t_offset = idx % trait_size;

    if (route.stop_count_ <= 32) {
      mc_update_route_smaller32<CriteriaConfig>(
          r_id, route, t_offset, prev_arrivals, arrivals, earliest_arrivals,
          target_stop_id, station_marks, tt);
    } else {
      mc_update_route_larger32<CriteriaConfig>(
          r_id, route, t_offset, prev_arrivals, arrivals, earliest_arrivals,
          target_stop_id, station_marks, tt);
    }
  }

  this_grid().sync();

  auto const store_size = (max_idx / 32) + 1;
  reset_store(route_marks, store_size);
}

template <typename CriteriaConfig>
__device__ void mc_update_footpaths_dev(device_memory const& device_mem,
                                        raptor_round const round_k,
                                        stop_id const target_stop_id,
                                        device_gpu_timetable const& tt) {
  time* const arrivals = device_mem.result_[round_k];

  // we must only copy the marked arrivals,
  // since an earlier raptor query might have used a footpath
  // to generate the current arrival, a new optimum from this value
  // would be generated using a double walk -> not correct!
  mc_copy_marked_arrivals<CriteriaConfig>(device_mem.footpaths_scratchpad_,
                                          arrivals, device_mem.station_marks_,
                                          tt);
  this_grid().sync();

  mc_update_footpaths_dev_scratch<CriteriaConfig>(
      device_mem.footpaths_scratchpad_, arrivals, device_mem.earliest_arrivals_,
      target_stop_id, device_mem.station_marks_, tt);
  this_grid().sync();
}

template <typename CriteriaConfig>
__device__ void mc_init_arrivals_dev(base_query const& query,
                                     device_memory const& device_mem,
                                     device_gpu_timetable const& tt) {
  auto const t_id = get_global_thread_id();

  auto const trait_size = CriteriaConfig::trait_size();
  if (t_id == 0) {
    auto const arr_idx = CriteriaConfig::get_arrival_idx(query.source_, 0);
    device_mem.result_[0][arr_idx] = query.source_time_begin_;
    mark(device_mem.station_marks_, arr_idx);
  }

  auto req_update_count = device_mem.additional_start_count_;
  auto global_stride = get_global_stride();
  for (auto idx = t_id; idx < req_update_count; idx += global_stride) {
    auto const add_start_idx = idx;

    auto const& add_start = device_mem.additional_starts_[add_start_idx];

    auto const add_start_time = query.source_time_begin_ + add_start.offset_;
    auto const add_start_arr_idx =
        CriteriaConfig::get_arrival_idx(add_start.s_id_, 0);
    bool updated = update_arrival(device_mem.result_[0], add_start_arr_idx,
                                  add_start_time);

    if (updated) {
      mark(device_mem.station_marks_, add_start_arr_idx);
    }
  }
}

template <typename CriteriaConfig>
__global__ void mc_gpu_raptor_kernel(base_query const query,
                                     device_memory const device_mem,
                                     device_gpu_timetable const tt) {
  mc_init_arrivals_dev<CriteriaConfig>(query, device_mem, tt);
  this_grid().sync();

  auto const trait_size = CriteriaConfig::trait_size();

  for (raptor_round round_k = 1; round_k < max_raptor_round; ++round_k) {

    //    if (get_global_thread_id() == 0)
    //      printf("Raptor Round %i\n==========\n", round_k);

    auto const t_id = get_global_thread_id();
    if (t_id < trait_size) {
      device_mem.any_station_marked_[t_id] = false;
    }
    if (t_id == 0) {
      *(device_mem.overall_station_marked_) = false;
    }
    this_grid().sync();

    mc_convert_station_to_route_marks(
        device_mem.station_marks_, device_mem.route_marks_,
        device_mem.any_station_marked_, device_mem.overall_station_marked_, tt,
        trait_size);
    this_grid().sync();


    auto const station_store_size = ((tt.stop_count_ * trait_size) / 32) + 1;
    reset_store(device_mem.station_marks_, station_store_size);
    this_grid().sync();

    if (!(*device_mem.overall_station_marked_)) {
      return;
    }

    time const* const prev_arrivals = device_mem.result_[round_k - 1];
    time* const arrivals = device_mem.result_[round_k];

    mc_update_routes_dev<CriteriaConfig>(
        prev_arrivals, arrivals, device_mem.earliest_arrivals_,
        device_mem.station_marks_, device_mem.route_marks_, query.target_, tt);

    this_grid().sync();

    mc_clear_dominated_arrivals<CriteriaConfig>(
        device_mem.stop_count_, device_mem.result_[round_k],
        device_mem.earliest_arrivals_, device_mem.station_marks_);
    this_grid().sync();

//    if(t_id == 0 && round_k == 2) {
//      printf("\nRoute Marks after:\n");
//      print_store(device_mem.station_marks_, tt.stop_count_ * trait_size,
//                  trait_size);
//    }
//    this_grid().sync();

    mc_update_footpaths_dev<CriteriaConfig>(device_mem, round_k, query.target_,
                                            tt);

    this_grid().sync();

    //    if (get_global_thread_id() == 0) {
    //      time t = invalid<time>;
    //      stop_id s = invalid<stop_id>;
    //      if (round_k == 1) {
    //        s = 8731;
    //      }
    //      if (round_k == 2) {
    //        s = 8754;
    //      }
    //      if (round_k == 3) {
    //        s = 7483;
    //      }
    //      if (round_k == 4) {
    //        s = 5946;
    //      }
    //      if (round_k == 5) {
    //        s = 11451;
    //      }
    //      if (round_k == 6) {
    //        s = 34195;
    //      }
    //      t = device_mem.result_[round_k][trait_size * s];
    //      printf("s_id: %i received time %i\n", s, t);
    //    }
    //    this_grid().sync();

    //    if(t_id == 0 && round_k == 1) {
    //      printf("Station Marks:\n");
    //      print_store(device_mem.station_marks_, tt.stop_count_ *
    //      trait_size,
    //                  trait_size);
    //    }
    //
    //    this_grid().sync();
  }
}

template <typename CriteriaConfig>
void invoke_mc_gpu_raptor(d_query const& dq) {
  void* kernel_args[] = {(void*)&dq, (void*)(dq.mem_->active_device_),
                         (void*)&dq.tt_};

  launch_kernel(mc_gpu_raptor_kernel<CriteriaConfig>, kernel_args,
                dq.mem_->context_, dq.mem_->context_.proc_stream_,
                dq.criteria_config_);
  cuda_check();

  cuda_sync_stream(dq.mem_->context_.proc_stream_);
  cuda_check();

  fetch_arrivals_async(dq, dq.mem_->context_.transfer_stream_);
  cuda_check();

  fetch_statistics_async(dq, dq.mem_->context_.transfer_stream_);
  cuda_check();

  cuda_sync_stream(dq.mem_->context_.transfer_stream_);
  cuda_check();
}

#define GENERATE_LAUNCH_CONFIG_FUNCTION(VAL, ACCESSOR)                        \
  template <>                                                                 \
  std::pair<dim3, dim3> get_mc_gpu_raptor_launch_parameters<VAL>(             \
      device_id const device_id, int32_t const concurrency_per_device) {      \
    hipSetDevice(device_id);                                                 \
    cuda_check();                                                             \
                                                                              \
    hipDeviceProp_t prop{};                                                    \
    hipGetDeviceProperties(&prop, device_id);                                \
    cuda_check();                                                             \
                                                                              \
    utl::verify(prop.warpSize == 32,                                          \
                "Warp Size must be 32! Otherwise the gRAPTOR algorithm will " \
                "not work.");                                                 \
                                                                              \
    int min_grid_size = 0;                                                    \
    int block_size = 0;                                                       \
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,           \
                                       mc_gpu_raptor_kernel<VAL>, 0, 0);      \
                                                                              \
    dim3 threads_per_block(prop.warpSize, block_size / prop.warpSize, 1);     \
    dim3 grid(min_grid_size / concurrency_per_device, 1, 1);                  \
                                                                              \
    return {threads_per_block, grid};                                         \
  }

#define MAKE_MC_GPU_RAPTOR_TEMPLATE_INSTANCE(VAL, ACCESSOR) \
  template void invoke_mc_gpu_raptor<VAL>(const d_query& dq);

#define MAKE_MC_INIT_ARRIVALS_TEMPLATE_INSTANCE(VAL, ACCESSOR) \
  template __device__ void mc_init_arrivals_dev<VAL>(          \
      base_query const&, device_memory const&, device_gpu_timetable const&);

#define MAKE_MC_UPDATE_FOOTPATHS_TEMPLATE_INSTANCE(VAL, ACCESSOR) \
  template __device__ void mc_update_footpaths_dev<VAL>(          \
      device_memory const&, raptor_round const, stop_id const,    \
      device_gpu_timetable const&);

#define MAKE_MC_UPDATE_ROUTES_TEMPLATE_INSTANCE(VAL, ACCESSOR)           \
  template __device__ void mc_update_routes_dev<VAL>(                    \
      time const* const, time* const, time* const, uint32_t*, uint32_t*, \
      stop_id const, device_gpu_timetable const&);

RAPTOR_CRITERIA_CONFIGS_WO_DEFAULT(GENERATE_LAUNCH_CONFIG_FUNCTION,
                                   raptor_criteria_config)

RAPTOR_CRITERIA_CONFIGS_WO_DEFAULT(MAKE_MC_INIT_ARRIVALS_TEMPLATE_INSTANCE, )
RAPTOR_CRITERIA_CONFIGS_WO_DEFAULT(MAKE_MC_UPDATE_FOOTPATHS_TEMPLATE_INSTANCE, )
RAPTOR_CRITERIA_CONFIGS_WO_DEFAULT(MAKE_MC_UPDATE_ROUTES_TEMPLATE_INSTANCE, )
RAPTOR_CRITERIA_CONFIGS_WO_DEFAULT(MAKE_MC_GPU_RAPTOR_TEMPLATE_INSTANCE, )

}  // namespace motis::raptor
