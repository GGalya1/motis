#include "hip/hip_runtime.h"
#include "motis/raptor/gpu/mc_gpu_raptor.cuh"

#include <iostream>

#include "motis/raptor/gpu/cuda_util.h"
#include "motis/raptor/gpu/gpu_mark_store.cuh"
#include "motis/raptor/gpu/raptor_utils.cuh"
#include "motis/raptor/gpu/update_arrivals.cuh"

#include "motis/raptor/criteria/configs.h"

#include "hip/hip_cooperative_groups.h"

namespace motis::raptor {

using namespace cooperative_groups;

// leader type must be unsigned 32bit
// no leader is a zero ballot vote (all 0) minus 1 => with underflow all 1's
constexpr unsigned int FULL_MASK = 0xFFFFffff;
constexpr unsigned int NO_LEADER = FULL_MASK;

template <typename CriteriaConfig>
__device__ void print_aggregate(CriteriaConfig const& d) {}

template <>
__device__ void print_aggregate<MaxTransferClass>(MaxTransferClass const& d) {
  printf("dep_off %i\tprev %i\tdep %i\ttt %i\tfast %i\tslow %i\tmtc %i\n",
         d._dep_offset, d._prev_arr, d._stop_dep, d._regular_tt, d._fast_tt,
         d._slow_tt, d.active_transfer_class_);
}

template <typename CriteriaConfig>
__device__ void mc_copy_marked_arrivals(time* const to, time const* const from,
                                        unsigned int* station_marks,
                                        device_gpu_timetable const& tt) {
  auto const global_stride = get_global_stride();

  auto arr_idx = get_global_thread_id();
  auto trait_size = CriteriaConfig::TRAITS_SIZE;
  auto max_arrival_idx = tt.stop_count_ * trait_size;
  for (; arr_idx < max_arrival_idx; arr_idx += global_stride) {
    // auto const stop_id = arr_idx / trait_size;

    // only copy the values for station + trait offset which are valid
    if (marked(station_marks, arr_idx) && valid(from[arr_idx])) {
      to[arr_idx] = from[arr_idx];
    } else {
      to[arr_idx] = invalid<time>;
    }
  }
}

template <typename CriteriaConfig>
__device__ void mc_copy_and_min_arrivals(time* const to, time* const from,
                                         device_gpu_timetable const& tt) {
  auto const global_stride = get_global_stride();

  auto arr_idx = get_global_thread_id();
  auto const max_arr_idx = tt.stop_count_ * CriteriaConfig::TRAITS_SIZE;
  for (; arr_idx < max_arr_idx; arr_idx += global_stride) {
    to[arr_idx] = min(from[arr_idx], to[arr_idx]);
  }
}

__device__ __forceinline__ unsigned get_criteria_propagation_mask(
    unsigned const leader, unsigned const stop_count) {
  auto const stops_to_update = stop_count - leader - 1;
  auto const mask = (1 << stops_to_update) - 1;
  return (mask << (leader + 1));
}

__device__ __forceinline__ unsigned get_last_departure_stop(
    unsigned criteria_mask) {
  unsigned const rev = __brev(criteria_mask);
  unsigned const last = 32 - __ffs(rev);
  return last;
}

__device__ __forceinline__ stop_offset get_closest_departure_sti(
    unsigned const ballot, stop_offset const arr_offset,
    uint32_t current_stage = 0,
    stop_offset const last_known_dep_stop_offset = invalid<stop_offset>) {
  assert(arr_offset > 0);
  // the shifted mask only contains the ballot results for all stops which are
  // located before the stop under investigation
  // it is then reversed so that the stop directly before the arrival stop
  // is at position 0 and the stop before that at position 1 and so forth
  // therefore, the index of the first 1 found in the mask represents the
  // distance in (measured in stops) to the departure stop
  // which can then be used to determine the departure offset
  auto const shift_by = 32 - arr_offset;
  auto const shifted = ballot << shift_by;
  auto const rev_shifted = __brev(shifted);
  auto const first_dep_idx = __ffs(rev_shifted);
  if (first_dep_idx > 0) {
    return (current_stage << 5) + arr_offset - first_dep_idx;
  } else {
    // this can only happen for routes with > 32 stops
    //  if there exists a departure stop in a stage before the current
    // then return the last known dep stop offset
    assert(valid(last_known_dep_stop_offset));
    return last_known_dep_stop_offset;
  }
}

template <typename CriteriaConfig>
__device__ __forceinline__ time get_earliest_arrival(
    const time* const earliest_arrivals, stop_id const target_stop_id,
    stop_id const current_stop_id, trait_id const write_to_offset) {

  arrival_id const current_idx =
      CriteriaConfig::get_arrival_idx(current_stop_id, write_to_offset);
  time const stop_ea = earliest_arrivals[current_idx];
  auto const target_arr_idx =
      CriteriaConfig::get_arrival_idx(target_stop_id, write_to_offset);
  time const target_ea = earliest_arrivals[target_arr_idx];
  return umin(stop_ea, target_ea);
}

template <typename CriteriaConfig>
__device__ void mc_update_route_larger32(
    route_id const r_id, gpu_route const route, trait_id const t_offset,
    time const* const prev_arrivals, time* const arrivals,
    time* const earliest_arrivals, stop_id const target_stop_id,
    uint32_t* station_marks, device_gpu_timetable const& tt,
    bool use_stop_satis) {

  auto const t_id = threadIdx.x;

  stop_id stop_id_t = invalid<stop_id>;
  time prev_arrival = invalid<time>;
  time stop_arrival = invalid<time>;
  time stop_departure = invalid<time>;

  CriteriaConfig aggregate{&route, t_offset};
  unsigned last_known_dep_stop = invalid<unsigned>;

  int active_stop_count = route.stop_count_;

  // this is ceil(stop_count / 32)
  int const stage_count = (route.stop_count_ + (32 - 1)) >> 5;
  int active_stage_count = stage_count;

  unsigned int leader = NO_LEADER;
  unsigned int any_arrival = 0;
  unsigned int criteria_mask = 0;

  for (int trip_offset = 0; trip_offset < route.trip_count_; ++trip_offset) {
    if (CriteriaConfig::USES_SHFL_CALC) aggregate.reset(trip_offset, t_offset);

    for (uint32_t current_stage = 0; current_stage < active_stage_count;
         ++current_stage) {
      if (!CriteriaConfig::USES_SHFL_CALC)
        aggregate.reset(trip_offset, t_offset);

      uint32_t stage_id = (current_stage << 5) + t_id;  // stage_id ^= stop_id

      // load the prev arrivals for the current stage
      if (stage_id < active_stop_count) {
        stop_id_t = tt.route_stops_[route.index_to_route_stops_ + stage_id];
        auto const stop_arr_idx =
            CriteriaConfig::get_arrival_idx(stop_id_t, t_offset);
        prev_arrival = prev_arrivals[stop_arr_idx];
      }

      any_arrival |= __any_sync(FULL_MASK, valid(prev_arrival));

      if (current_stage == active_stage_count - 1 && !any_arrival) {
        return;
      }

      if (!any_arrival) {
        continue;
      }

      // load the stop times for the current stage
      bool departure_feasible = false;
      if (stage_id < active_stop_count) {
        auto const st_idx = route.index_to_stop_times_ +
                            (trip_offset * route.stop_count_) + stage_id;
        stop_departure = tt.stop_departures_[st_idx];
        departure_feasible = aggregate.check_and_set_departure_stop(
            tt, stage_id, stop_id_t, prev_arrival, stop_departure);
      }

      // TODO adapted for TT
      // get the current stage leader
      unsigned int ballot = __ballot_sync(
          FULL_MASK, (stage_id < active_stop_count) && valid(prev_arrival) &&
                         valid(stop_departure) && departure_feasible);

      // index of first possible departure station on this stage
      leader = __ffs(ballot) - 1;

      auto const stop_count_on_stage =
          active_stop_count < ((current_stage + 1) << 5)
              ? (active_stop_count - (current_stage << 5))
              : 32;

      if (valid(last_known_dep_stop)) {
        // no leader in current stags; though maybe there is a leader in the
        //   previous stage
        //        printf("Active Stop count %i;\ton stage: %i\n",
        //        stop_count_on_stage, current_stage);
        criteria_mask = (1 << stop_count_on_stage) - 1;
      } else if (leader != NO_LEADER) {
        criteria_mask =
            get_criteria_propagation_mask(leader, stop_count_on_stage);
      } else {
        // no departure stop up to now
        continue;
      }

      if (leader > 0 && valid(last_known_dep_stop) &&
          CriteriaConfig::USES_SHFL_CALC) {
        // in a case where there is no departure location in this stage
        // or the departure location (leader) is after this stop
        // and there is a valid known departure stop from teh previous stage
        // carry over the aggregate from the last stop in the previous stage
        // to the first stop in this stage
        unsigned carry_mask = (1 << 31) | 1;
        aggregate.carry_to_next_stage(carry_mask);
      }

      auto const has_carry_value = t_id == 0 && valid(last_known_dep_stop);
      if (!has_carry_value) {

        // don't reset if this has a carry value even if it is a departure
        // station
        //  as arrival time might be improved through the carry value
        aggregate.reset(trip_offset, t_offset);
      }

      if (leader != NO_LEADER) {
        // adjust the determined departure location to the current stage
        leader += current_stage << 5;
      }

      // update this stage if there is a leader or a known dep stop from
      // one of the previous stages
      bool wrote_satis_time = false;
      if ((leader != NO_LEADER || valid(last_known_dep_stop)) &&
          stage_id < active_stop_count) {

        if ((leader != NO_LEADER && stage_id > leader) ||
            (valid(last_known_dep_stop) && stage_id > last_known_dep_stop)) {
          auto const first_sti =
              route.index_to_stop_times_ + (trip_offset * route.stop_count_);
          auto const st_idx = first_sti + stage_id;
          stop_arrival = tt.stop_arrivals_[st_idx];

          if (CriteriaConfig::USES_SHFL_CALC) {
            // is_departure_stop is local to the current stage
            auto const is_departure_stop = (((1 << t_id) & ballot) >> t_id);

            aggregate.update_from_stop(tt, t_id, st_idx);

            // propagate the additional criteria attributes
            for (uint32_t idx = __ffs(criteria_mask); idx < stop_count_on_stage;
                 ++idx) {
              // internally uses __shfl_up_sync to propagate the criteria values
              //  along the traits while allowing for max/min/sum operations
              aggregate.propagate_along_warp(
                  criteria_mask, is_departure_stop,
                  idx <= t_id
                      // prevent write update if this has carry value
                      && !has_carry_value);
            }
          } else {

            auto const dep_offset = get_closest_departure_sti(
                ballot, t_id, current_stage, last_known_dep_stop);
            auto const dep_sti = first_sti + dep_offset;
            aggregate.calculate(tt, prev_arrivals, dep_sti, st_idx);
          }

          auto const write_to_offset = aggregate.get_write_to_trait_id();

          if (valid(write_to_offset)) {
            auto const earliest_arrival = get_earliest_arrival<CriteriaConfig>(
                earliest_arrivals, target_stop_id, stop_id_t, write_to_offset);

            if (stop_arrival < earliest_arrival) {
              auto const write_to_arr_idx =
                  CriteriaConfig::get_arrival_idx(stop_id_t, write_to_offset);
              bool updated =
                  update_arrival(arrivals, write_to_arr_idx, stop_arrival);
              if (updated) {
                wrote_satis_time = aggregate.is_satisfied(t_offset);

                update_arrival(earliest_arrivals, write_to_arr_idx,
                               stop_arrival);
                mark(station_marks, write_to_arr_idx);
              }
            }
          }
        }
      }

      if (leader != NO_LEADER) {
        if (current_stage == active_stage_count - 1 && use_stop_satis) {
          // at the last stage check the stop satisfaction and reduce asc if
          // possible
          time satisfied_ea = invalid<time>;
          if ((1 << t_id) & criteria_mask) {
            auto satisfied_arr_idx =
                CriteriaConfig::get_arrival_idx(stop_id_t, t_offset);
            satisfied_ea = get_earliest_arrival<CriteriaConfig>(
                earliest_arrivals, stop_id_t, target_stop_id, t_offset);
          }

          auto satisfied_ballot = __ballot_sync(
              FULL_MASK,
              stage_id < active_stop_count &&
                  (wrote_satis_time ||
                   (valid(stop_arrival) && satisfied_ea <= stop_arrival)));

          auto stage_asc = active_stop_count - (current_stage << 5);
          auto const helper_mask =
              stage_asc < 32 ? __brev((1 << (32 - stage_asc)) - 1) : 0;
          auto const inverted_ballot = ~(satisfied_ballot | helper_mask);
          auto leading_zero_count = __clz(inverted_ballot);
          //          auto const init_lzc = leading_zero_count;
          leading_zero_count -= (32 - stage_asc);
          active_stop_count -= leading_zero_count;

          //          auto const init_stage_asc = stage_asc;
          stage_asc -= leading_zero_count;

          if (stage_asc == 0) {
            active_stage_count -= 1;
          }

        } else {
          // there is a leader in the current stage; therefore safe the last
          // possible departure stop for updates to the next stage
          last_known_dep_stop = get_closest_departure_sti(
              ballot, 32, current_stage, last_known_dep_stop);
          //          last_known_dep_stop += current_stage << 5;
        }
      }
    }
  }
}

template <typename CriteriaConfig>
__device__ void mc_update_route_smaller32(
    route_id const r_id, gpu_route const route, trait_id const t_offset,
    time const* const prev_arrivals, time* const arrivals,
    time* const earliest_arrivals, stop_id const target_stop_id,
    uint32_t* station_marks, device_gpu_timetable const& tt,
    bool use_stop_satis) {

  auto const t_id = threadIdx.x;

  stop_id s_id = invalid<stop_id>;
  time prev_arrival = invalid<time>;
  time stop_arrival = invalid<time>;
  time stop_departure = invalid<time>;

  CriteriaConfig aggregate{&route, t_offset};

  unsigned leader = route.stop_count_;
  unsigned int active_stop_count = route.stop_count_;

  if (t_id < active_stop_count) {
    s_id = tt.route_stops_[route.index_to_route_stops_ + t_id];
    auto const stop_arr_idx = CriteriaConfig::get_arrival_idx(s_id, t_offset);
    prev_arrival = prev_arrivals[stop_arr_idx];
  }

  // we skip updates if there is no feasible departure station
  //  on this route with the given trait offset
  if (!__any_sync(FULL_MASK, valid(prev_arrivals))) {
    return;
  }

  for (trip_id trip_offset = 0; trip_offset < route.trip_count_;
       ++trip_offset) {
    aggregate.reset(trip_offset, t_offset);

    bool departure_feasible = false;
    if (t_id < active_stop_count) {
      auto const st_index =
          route.index_to_stop_times_ + (trip_offset * route.stop_count_) + t_id;
      stop_departure = tt.stop_departures_[st_index];
      departure_feasible = aggregate.check_and_set_departure_stop(
          tt, t_id, s_id, prev_arrival, stop_departure);
    }

    // TODO adapted for TT
    unsigned ballot = __ballot_sync(
        FULL_MASK, (t_id < active_stop_count) && valid(prev_arrival) &&
                       valid(stop_departure) && departure_feasible);

    // index of the first departure location on route
    leader = __ffs(ballot) - 1;

    if (leader == NO_LEADER) continue;  // No feasible departure on this trip

    unsigned criteria_mask =
        get_criteria_propagation_mask(leader, active_stop_count);

    bool wrote_satis_time = false;
    if (t_id > leader && t_id < active_stop_count) {
      auto const first_sti =
          route.index_to_stop_times_ + (trip_offset * route.stop_count_);
      auto const st_index = first_sti + t_id;

      stop_arrival = tt.stop_arrivals_[st_index];

      if (CriteriaConfig::USES_SHFL_CALC) {
        auto const is_departure_stop = (((1 << t_id) & ballot) >> t_id);
        aggregate.update_from_stop(tt, t_id /* == stop offset */, st_index);

        // propagate the additional criteria attributes
        for (uint32_t idx = leader + 1; idx < active_stop_count; ++idx) {
          // internally uses __shfl_up_sync to propagate the criteria values
          //  along the traits while allowing for max/min/sum operations
          aggregate.propagate_along_warp(criteria_mask, is_departure_stop,
                                         idx <= t_id);
        }
      } else {
        auto const dep_offset = get_closest_departure_sti(ballot, t_id);
        auto const dep_sti = first_sti + dep_offset;
        aggregate.calculate(tt, prev_arrivals, dep_sti, st_index);
      }

      auto const write_to_offset = aggregate.get_write_to_trait_id();
      if (valid(write_to_offset) && t_id > leader) {

        // Note: Earliest Arrival may, when reaching this point not be the
        //       'earliest arrival' at this stop, but it gives a sufficient
        //       upper bound and allows preventing arrival time which are the
        //       same as for one round earlier
        auto const earliest_arrival = get_earliest_arrival<CriteriaConfig>(
            earliest_arrivals, target_stop_id, s_id, write_to_offset);

        if (stop_arrival < earliest_arrival) {
          auto const write_to_idx =
              CriteriaConfig::get_arrival_idx(s_id, write_to_offset);
          bool updated = update_arrival(arrivals, write_to_idx, stop_arrival);
          if (updated) {
            wrote_satis_time = aggregate.is_satisfied(t_offset);

            update_arrival(earliest_arrivals, write_to_idx, stop_arrival);
            mark(station_marks, write_to_idx);
          }
        }
      }
    }

    // check if stops on route are satisfied
    if (leader != NO_LEADER && use_stop_satis) {
      time satisfied_ea = invalid<time>;
      if ((1 << t_id) & criteria_mask) {
        auto satisfied_arr_idx =
            CriteriaConfig::get_arrival_idx(s_id, t_offset);
        satisfied_ea = get_earliest_arrival<CriteriaConfig>(
            earliest_arrivals, s_id, target_stop_id, t_offset);
      }

      auto const satisfied_ballot = __ballot_sync(
          FULL_MASK, t_id < active_stop_count &&
                         (wrote_satis_time || (valid(stop_arrival) &&
                                               satisfied_ea <= stop_arrival)));

      // not satisfied yet but there's a chance we can reduce
      // the number of stops to be scanned on the next trip
      auto const helper_mask = __brev((1 << (32 - active_stop_count)) - 1);
      auto inverted_ballot = ~(satisfied_ballot | helper_mask);
      auto leading_zero_count = __clz(inverted_ballot);
      //      auto const initial_clz = leading_zero_count;
      leading_zero_count -= (32 - active_stop_count);

      //      auto const initial_acs = active_stop_count;
      active_stop_count -= leading_zero_count;

      // if every stop is satisfied we can skip further updates
      if ((1 << route.stop_count_) - 1 == satisfied_ballot) {
        break;
      }
    }
    leader = NO_LEADER;
  }
}

template <typename CriteriaConfig>
__device__ void mc_update_footpaths_dev_scratch(
    time const* const read_arrivals, time* const write_arrivals,
    time* const earliest_arrivals, stop_id const target_stop_id,
    uint32_t* station_marks, device_gpu_timetable const& tt) {

  auto const global_stride = get_global_stride();

  auto arrival_idx = get_global_thread_id();
  auto const trait_size = CriteriaConfig::TRAITS_SIZE;
  auto const max_arr_idx = tt.footpath_count_ * trait_size;
  auto const target_arr_idx = CriteriaConfig::get_arrival_idx(target_stop_id);

  for (; arrival_idx < max_arr_idx; arrival_idx += global_stride) {
    auto const foot_idx = arrival_idx / trait_size;
    auto const t_offset = arrival_idx % trait_size;

    auto const footpath = tt.footpaths_[foot_idx];

    auto const from_arrival_idx =
        CriteriaConfig::get_arrival_idx(footpath.from_, t_offset);
    auto const to_arrival_idx =
        CriteriaConfig::get_arrival_idx(footpath.to_, t_offset);

    time const from_arrival = read_arrivals[from_arrival_idx];
    if (valid(from_arrival)) {
      time const new_arrival = from_arrival + footpath.duration_;

      // this give potentially just an upper bound and not the real
      //  earliest arrival value at the time the update is written
      time const to_stop_ea = earliest_arrivals[to_arrival_idx];
      time const target_ea = earliest_arrivals[target_arr_idx + t_offset];
      time const earliest_arrival = umin(to_stop_ea, target_ea);

      if (marked(station_marks, from_arrival_idx) &&
          new_arrival < earliest_arrival) {
        bool updated =
            update_arrival(write_arrivals, to_arrival_idx, new_arrival);
        if (updated) {
          update_arrival(earliest_arrivals, to_arrival_idx, new_arrival);
          mark(station_marks, to_arrival_idx);
        }
      }
    }
  }
}

template <typename CriteriaConfig>
__device__ void perform_arrival_sweeping(stop_id const stop_count,
                                         time* const arrivals, time* const ea,
                                         uint32_t* station_marks) {
  auto const global_stride = get_global_stride();

  auto const trait_size = CriteriaConfig::TRAITS_SIZE;
  auto const block_size = CriteriaConfig::SWEEP_BLOCK_SIZE;

  if (block_size == 1) return;

  auto s_id = get_global_thread_id();
  // one thread scans all arrivals on one stop
  for (; s_id < stop_count; s_id += global_stride) {
    for (trait_id t_offset = 0; t_offset < trait_size; t_offset += block_size) {

      time min_at_stop = arrivals[trait_size * s_id + t_offset];
      for (trait_id block_off = t_offset + 1; block_off < t_offset + block_size;
           ++block_off) {
        arrival_id const arr_idx =
            CriteriaConfig::get_arrival_idx(s_id, block_off);
        time const current = arrivals[arr_idx];
        // if the value is larger or equal than the minimum we can prune it
        //   because it is dominated by the minimum on the earliest trait offset
        if (valid(min_at_stop) && valid(current) && min_at_stop <= current) {
          arrivals[arr_idx] = invalid<time>;
          unmark(station_marks, arr_idx);
          if (min_at_stop <= ea[arr_idx]) {
            ea[arr_idx] = invalid<time>;
          }
        } else if (current < min_at_stop) {
          // a higher t_offset has a better value; remember the larger value
          //  to again check higher t_offsets against it
          min_at_stop = current;
        }
      }
    }
  }
}

template <typename CriteriaConfig>
__device__ void mc_update_routes_dev(device_memory const& device_mem,
                                     raptor_round const round_k,
                                     stop_id const target_stop_id,
                                     device_gpu_timetable const& tt,
                                     bool use_stop_satis) {

  time const* const prev_arrivals = device_mem.result_[round_k - 1];
  time* const arrivals = device_mem.result_[round_k];
  uint32_t* route_marks = device_mem.route_marks_;
  uint32_t* station_marks = device_mem.station_marks_;
  time* const earliest_arrivals = device_mem.earliest_arrivals_;

  // blockDim.x = 32; blockDim.y = 32; gridDim.x =
  // 6; => Stride = 32*6 => 192
  auto const stride = blockDim.y * gridDim.x;
  // threadIdx.y = 1..32 + (blockDim.y = 32 * blockIdx.x = 1..6)
  auto const start_idx = threadIdx.y + (blockDim.y * blockIdx.x);

  uint32_t route_count = 0;

  auto const trait_size = CriteriaConfig::TRAITS_SIZE;
  auto const max_idx = tt.route_count_ * trait_size;
  for (auto idx = start_idx; idx < max_idx; idx += stride) {
    if (!marked(route_marks, idx)) {
      continue;
    }

    if (threadIdx.x == 0) {
      ++route_count;
    }

    auto const r_id = idx / trait_size;
    auto const route = tt.routes_[r_id];
    auto const t_offset = idx % trait_size;

    if (route.stop_count_ <= 32) {
      mc_update_route_smaller32<CriteriaConfig>(
          r_id, route, t_offset, prev_arrivals, arrivals, earliest_arrivals,
          target_stop_id, station_marks, tt, use_stop_satis);
    } else {
      mc_update_route_larger32<CriteriaConfig>(
          r_id, route, t_offset, prev_arrivals, arrivals, earliest_arrivals,
          target_stop_id, station_marks, tt, use_stop_satis);
    }
  }

  this_grid().sync();

  if (threadIdx.x == 0) {
    if (round_k == 1)
      atomicAdd(&device_mem.stats_->scanned_routes_1_, route_count);
    if (round_k == 2)
      atomicAdd(&device_mem.stats_->scanned_routes_2_, route_count);
    if (round_k == 3)
      atomicAdd(&device_mem.stats_->scanned_routes_3_, route_count);
    if (round_k == 4)
      atomicAdd(&device_mem.stats_->scanned_routes_4_, route_count);
    if (round_k == 5)
      atomicAdd(&device_mem.stats_->scanned_routes_5_, route_count);
    if (round_k == 6)
      atomicAdd(&device_mem.stats_->scanned_routes_6_, route_count);
    if (round_k == 7)
      atomicAdd(&device_mem.stats_->scanned_routes_7_, route_count);
  }

  auto const store_size = (max_idx / 32) + 1;
  reset_store(route_marks, store_size);
}

template <typename CriteriaConfig>
__device__ void mc_update_footpaths_dev(device_memory const& device_mem,
                                        raptor_round const round_k,
                                        stop_id const target_stop_id,
                                        device_gpu_timetable const& tt) {
  time* const arrivals = device_mem.result_[round_k];

  // we must only copy the marked arrivals,
  // since an earlier raptor query might have used a footpath
  // to generate the current arrival, a new optimum from this value
  // would be generated using a double walk -> not correct!
  mc_copy_marked_arrivals<CriteriaConfig>(device_mem.footpaths_scratchpad_,
                                          arrivals, device_mem.station_marks_,
                                          tt);
  this_grid().sync();

  mc_update_footpaths_dev_scratch<CriteriaConfig>(
      device_mem.footpaths_scratchpad_, arrivals, device_mem.earliest_arrivals_,
      target_stop_id, device_mem.station_marks_, tt);
  this_grid().sync();
}

template <typename CriteriaConfig>
__device__ void mc_init_arrivals_dev(base_query const& query,
                                     device_memory const& device_mem,
                                     device_gpu_timetable const& tt) {
  auto const t_id = get_global_thread_id();
  auto const trait_size = CriteriaConfig::TRAITS_SIZE;
  auto const sweep_block_size = CriteriaConfig::SWEEP_BLOCK_SIZE;

  auto write_to_trait_blocks = [&](time* const arrivals, stop_id const s_id,
                                   time const source_time) {
    auto const first_arr_idx = s_id * trait_size;
    auto const last_arr_idx = first_arr_idx + trait_size;

    for (auto arr_idx = first_arr_idx; arr_idx < last_arr_idx;
         arr_idx += sweep_block_size) {
      auto const t_offset = arr_idx - first_arr_idx;
      auto const trans_time =
          CriteriaConfig::get_transfer_time(tt, t_offset, s_id);
      auto const arriv_time = source_time - trans_time;

      bool updated = update_arrival(arrivals, arr_idx, arriv_time);
      if (updated) {
        mark(device_mem.station_marks_, arr_idx);
      }
    }
  };

  // TODO adapted for TT
  if (t_id == 0) {
    write_to_trait_blocks(device_mem.result_[0], query.source_,
                          query.source_time_begin_);
  }

  auto req_update_count = device_mem.additional_start_count_;
  auto global_stride = get_global_stride();
  for (auto add_start_idx = t_id; add_start_idx < req_update_count;
       add_start_idx += global_stride) {

    auto const& add_start = device_mem.additional_starts_[add_start_idx];

    auto const add_start_time = query.source_time_begin_ + add_start.offset_;

    write_to_trait_blocks(device_mem.result_[0], add_start.s_id_,
                          add_start_time);
  }
}

template <typename CriteriaConfig>
__global__ void mc_gpu_raptor_kernel(base_query const query,
                                     device_memory const device_mem,
                                     device_gpu_timetable const tt,
                                     bool use_arr_sweep,
                                     bool use_stop_satis) {
  auto const trait_size = CriteriaConfig::TRAITS_SIZE;
  auto const t_id = get_global_thread_id();

  mc_init_arrivals_dev<CriteriaConfig>(query, device_mem, tt);
  this_grid().sync();

  for (raptor_round round_k = 1; round_k < max_raptor_round; ++round_k) {
    if (t_id < trait_size) {
      device_mem.any_station_marked_[t_id] = false;
    }
    if (t_id == 0) {
      *(device_mem.overall_station_marked_) = false;
    }
    this_grid().sync();

    mc_convert_station_to_route_marks(
        device_mem.station_marks_, device_mem.route_marks_,
        device_mem.any_station_marked_, device_mem.overall_station_marked_, tt,
        trait_size);
    this_grid().sync();

    auto const station_store_size = ((tt.stop_count_ * trait_size) / 32) + 1;
    reset_store(device_mem.station_marks_, station_store_size);
    this_grid().sync();

    if (!(*device_mem.overall_station_marked_)) {
      return;
    }

    mc_update_routes_dev<CriteriaConfig>(device_mem, round_k, query.target_,
                                         tt, use_stop_satis);

    this_grid().sync();

    if (use_arr_sweep) {
      perform_arrival_sweeping<CriteriaConfig>(
          device_mem.stop_count_, device_mem.result_[round_k],
          device_mem.earliest_arrivals_, device_mem.station_marks_);
      this_grid().sync();
    }

    mc_update_footpaths_dev<CriteriaConfig>(device_mem, round_k, query.target_,
                                            tt);
    this_grid().sync();

    if (t_id == 0) {
      device_mem.stats_->number_of_rounds_ = round_k;
    }
  }

  if(t_id == 0) {
    device_mem.stats_->total_scanned_routes_ =
        device_mem.stats_->scanned_routes_1_ +
        device_mem.stats_->scanned_routes_2_ +
        device_mem.stats_->scanned_routes_3_ +
        device_mem.stats_->scanned_routes_4_ +
        device_mem.stats_->scanned_routes_5_ +
        device_mem.stats_->scanned_routes_6_ +
        device_mem.stats_->scanned_routes_7_;
  }
}

template <typename CriteriaConfig>
void invoke_mc_gpu_raptor(d_query const& dq) {
  void* kernel_args[] = {(void*)&dq, (void*)(dq.mem_->active_device_),
                         (void*)&dq.tt_,
                         (void*)&dq.use_arr_sweep_, (void*)&dq.use_stop_satis_
  };

  launch_kernel(mc_gpu_raptor_kernel<CriteriaConfig>, kernel_args,
                dq.mem_->context_, dq.mem_->context_.proc_stream_,
                dq.criteria_config_);
  cuda_check();
  fflush(stdout);

  cuda_sync_stream(dq.mem_->context_.proc_stream_);
  cuda_check();

  fetch_arrivals_async(dq, dq.mem_->context_.transfer_stream_);
  cuda_check();

  fetch_statistics_async(dq, dq.mem_->context_.transfer_stream_);
  cuda_check();

  cuda_sync_stream(dq.mem_->context_.transfer_stream_);
  cuda_check();
}

#define GENERATE_LAUNCH_CONFIG_FUNCTION(VAL, ACCESSOR)                        \
  template <>                                                                 \
  std::pair<dim3, dim3> get_mc_gpu_raptor_launch_parameters<VAL>(             \
      device_id const device_id, int32_t const concurrency_per_device) {      \
    hipSetDevice(device_id);                                                 \
    cuda_check();                                                             \
                                                                              \
    hipDeviceProp_t prop{};                                                    \
    hipGetDeviceProperties(&prop, device_id);                                \
    cuda_check();                                                             \
                                                                              \
    utl::verify(prop.warpSize == 32,                                          \
                "Warp Size must be 32! Otherwise the gRAPTOR algorithm will " \
                "not work.");                                                 \
                                                                              \
    int min_grid_size = 0;                                                    \
    int block_size = 0;                                                       \
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,           \
                                       mc_gpu_raptor_kernel<VAL>, 0, 0);      \
                                                                              \
    dim3 threads_per_block(prop.warpSize, block_size / prop.warpSize, 1);     \
    dim3 grid(min_grid_size / concurrency_per_device, 1, 1);                  \
                                                                              \
    return {threads_per_block, grid};                                         \
  }

#define MAKE_MC_GPU_RAPTOR_TEMPLATE_INSTANCE(VAL, ACCESSOR) \
  template void invoke_mc_gpu_raptor<VAL>(const d_query& dq);

#define MAKE_MC_INIT_ARRIVALS_TEMPLATE_INSTANCE(VAL, ACCESSOR) \
  template __device__ void mc_init_arrivals_dev<VAL>(          \
      base_query const&, device_memory const&, device_gpu_timetable const&);

#define MAKE_MC_UPDATE_FOOTPATHS_TEMPLATE_INSTANCE(VAL, ACCESSOR) \
  template __device__ void mc_update_footpaths_dev<VAL>(          \
      device_memory const&, raptor_round const, stop_id const,    \
      device_gpu_timetable const&);

#define MAKE_MC_UPDATE_ROUTES_TEMPLATE_INSTANCE(VAL, ACCESSOR) \
  template __device__ void mc_update_routes_dev<VAL>(          \
      device_memory const&, raptor_round const, stop_id const, \
      device_gpu_timetable const&, bool use_stop_satis);

RAPTOR_CRITERIA_CONFIGS_WO_DEFAULT(GENERATE_LAUNCH_CONFIG_FUNCTION,
                                   raptor_criteria_config)

RAPTOR_CRITERIA_CONFIGS_WO_DEFAULT(MAKE_MC_INIT_ARRIVALS_TEMPLATE_INSTANCE, )
RAPTOR_CRITERIA_CONFIGS_WO_DEFAULT(MAKE_MC_UPDATE_FOOTPATHS_TEMPLATE_INSTANCE, )
RAPTOR_CRITERIA_CONFIGS_WO_DEFAULT(MAKE_MC_UPDATE_ROUTES_TEMPLATE_INSTANCE, )
RAPTOR_CRITERIA_CONFIGS_WO_DEFAULT(MAKE_MC_GPU_RAPTOR_TEMPLATE_INSTANCE, )

}  // namespace motis::raptor
